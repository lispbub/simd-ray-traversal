#include "hip/hip_runtime.h"
/*
*  Copyright (c) 2009-2011, NVIDIA Corporation
*  All rights reserved.
*
*  Redistribution and use in source and binary forms, with or without
*  modification, are permitted provided that the following conditions are met:
*      * Redistributions of source code must retain the above copyright
*        notice, this list of conditions and the following disclaimer.
*      * Redistributions in binary form must reproduce the above copyright
*        notice, this list of conditions and the following disclaimer in the
*        documentation and/or other materials provided with the distribution.
*      * Neither the name of NVIDIA Corporation nor the
*        names of its contributors may be used to endorse or promote products
*        derived from this software without specific prior written permission.
*
*  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
*  ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
*  WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
*  DISCLAIMED. IN NO EVENT SHALL <COPYRIGHT HOLDER> BE LIABLE FOR ANY
*  DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
*  (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
*  LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND
*  ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
*  (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
*  SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

#include "CudaTracerKernels.hpp"

//------------------------------------------------------------------------

#define STACK_SIZE              24          // Size of the traversal stack in local memory.
#define BLOCK_HEIGHT             2
#define NUM_SUBWARPS            16
#define SUBWARP_WIDTH            2
#define CUDA_INF_F __int_as_float(0x7f800000)

extern "C" __device__ int g_warpCounter;    // Work counter for persistent threads.

//------------------------------------------------------------------------

extern "C" __global__ void queryConfig(void)
{
	g_config.bvhLayout = BVHLayout_Bin;
	g_config.blockWidth = 32;
	g_config.blockHeight = BLOCK_HEIGHT;
	g_config.usePersistentThreads = 1;
	g_config.desiredWarps = 960;
}

//------------------------------------------------------------------------

TRACE_FUNC
{

// Traversal stack in CUDA shared memory.
__shared__ volatile int traversalStack[NUM_SUBWARPS][BLOCK_HEIGHT][STACK_SIZE];

// Live state during traversal, stored in registers.
float   origx, origy, origz;            // Ray origin.
float   tmin;
float   hitT;
int     rayidx;
int     stackPtr = -1;
float   oodx, oody, oodz;
float   dirx, diry, dirz;
float   idirx, idiry, idirz;
int     hitIndex = -1;

const int offset = (threadIdx.x & 0x00000001);
const int subwarp = (threadIdx.x >> 1);
const int subwarp_mask = (0x00000003 << (threadIdx.x & 0xfffffffe));

// Initialize persistent threads.
// Persistent threads: fetch and process rays in a loop.
do
{
	// Fetch new rays from the global pool using lane 0.
	if (stackPtr < 0)
	{
		if (threadIdx.x == 0)
			rayidx = atomicAdd(&g_warpCounter, NUM_SUBWARPS);
		rayidx = __shfl(rayidx, 0) + subwarp;

		if (rayidx >= numRays)
			break;

		// Fetch ray.

		const float4 o = FETCH_GLOBAL(rays, rayidx * 2 + 0, float4);
		const float4 d = FETCH_GLOBAL(rays, rayidx * 2 + 1, float4);
		origx = o.x;
		origy = o.y;
		origz = o.z;
		tmin = o.w;
		dirx = d.x;
		diry = d.y;
		dirz = d.z;
		hitT = d.w;
		const float ooeps = exp2f(-80.0f); // Avoid div by zero.
		idirx = 1.0f / (fabsf(d.x) > ooeps ? d.x : copysignf(ooeps, d.x));
		idiry = 1.0f / (fabsf(d.y) > ooeps ? d.y : copysignf(ooeps, d.y));
		idirz = 1.0f / (fabsf(d.z) > ooeps ? d.z : copysignf(ooeps, d.z));
		oodx = origx * idirx;
		oody = origy * idiry;
		oodz = origz * idirz;
		// Setup traversal.
		stackPtr = 0;
		hitIndex = -1;
		if (!offset)
			traversalStack[subwarp][threadIdx.y][0] = 0;

	}

	// Traversal loop.

	while (stackPtr >= 0)
	{
	
		const int curr = traversalStack[subwarp][threadIdx.y][stackPtr--];
		if (curr >= 0)
		{
			// Fetch AABBs of the two child nodes.
			const float4 xy = tex1Dfetch(t_nodesA, curr + offset);
			const float4 zi = tex1Dfetch(t_nodesA, curr + SUBWARP_WIDTH + offset);

			// Intersect the ray against the child nodes.
			const float c0lox = xy.x * idirx - oodx;
			const float c0hix = xy.y * idirx - oodx;
			const float c0loy = xy.z * idiry - oody;
			const float c0hiy = xy.w * idiry - oody;		
			const float c0loz = zi.x * idirz - oodz;
			const float c0hiz = zi.y * idirz - oodz;
			int link = float_as_int(zi.z);

			const float c0min = spanBeginKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, tmin);
			const bool hit = c0min <= spanEndKepler(c0lox, c0hix, c0loy, c0hiy, c0loz, c0hiz, hitT);
			

			float dist = hit ? c0min : CUDA_INF_F;
			const int  hits = __popc(__ballot(hit)&subwarp_mask);
			if (!hits) continue;
			stackPtr += hits;


			//sort hits
			swap(dist, link, 0x01, bfe(threadIdx.x, 0));

			if (dist < CUDA_INF_F)
				traversalStack[subwarp][threadIdx.y][stackPtr - offset] = link;
		}
		else {

			// Load triangle

			int triAddr = ~curr + offset * 3;
			const float4 v00 = tex1Dfetch(t_trisA, triAddr + 0);
			const float4 v11 = tex1Dfetch(t_trisA, triAddr + 1);
			const float4 v22 = tex1Dfetch(t_trisA, triAddr + 2);

			const float Oz = v00.w - origx*v00.x - origy*v00.y - origz*v00.z;
			const float invDz = 1.0f / (dirx*v00.x + diry*v00.y + dirz*v00.z);
			float t = Oz * invDz;
			bool hit = false;

			if (t > tmin && t < hitT)
			{
				// Compute and check barycentric u.

				const float Ox = v11.w + origx*v11.x + origy*v11.y + origz*v11.z;
				const float Dx = dirx*v11.x + diry*v11.y + dirz*v11.z;
				const float u = Ox + t*Dx;

				if (u >= 0.0f)
				{
					// Compute and check barycentric v.

					const float Oy = v22.w + origx*v22.x + origy*v22.y + origz*v22.z;
					const float Dy = dirx*v22.x + diry*v22.y + dirz*v22.z;
					const float v = Oy + t*Dy;

					if (v >= 0.0f && u + v <= 1.0f)
					{
						// Record intersection.
						// Closest intersection not required => terminate.

						hit = true;
					}
				}
			}

			// Sort triangles
	
			const int hits = __ballot(hit)&subwarp_mask;
			if (!hits) continue;

			t = hit ? t : CUDA_INF_F;

			{
				const float tmp_t = __shfl_xor(t, 1);
				const int tmp_addr = __shfl_xor(triAddr, 1);

				triAddr = tmp_t < t ? tmp_addr : triAddr;
				t = fminf(t, tmp_t);
			}

			hitIndex = triAddr;
			hitT = t;

			if (anyHit && hits)
			{
				stackPtr = -1;
				break;
			}

		}
	}

	if (offset == 0) {
		if (hitIndex == -1) { STORE_RESULT(rayidx, -1, hitT); }
		else { STORE_RESULT(rayidx, FETCH_TEXTURE(triIndices, hitIndex, int), hitT); }
	}

} while (true);
}

//------------------------------------------------------------------------
